#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include ""
#include "hip/hip_runtime.h"
#include "bmp.cuh"

__global__ void cudaFunc(unsigned char* data, int size, int Nt, int n, int* count) {
	for (int i = size - 1 - 3 * threadIdx.x; i >= 0; i -= 3 * Nt) {
		
		unsigned char red = data[i];
		unsigned char green = data[i - 1];
		unsigned char blue = data[i - 2];
		if (red * green * blue < n) {
			count[threadIdx.x] +=1;
		}
	}
}

int main(int argc, char *argv[]) {

	const char* name = "img01.bmp";
	BMP_File* bmp = loadBMP(name);
	const int NT = 7; //���������� �����
	int N = 1000;

	//�������������� ������� �� ������� ������
	int *count = (int*)malloc(sizeof(int) * NT);
	for (int i = 0; i < NT; i++) {
		count[i] = 0;
	}

	//�������� ������ �� cpu ��� ��������
	unsigned char *imgData;
	imgData = (unsigned char*)malloc(sizeof(unsigned char) * bmp->dh.data_size);
	imgData = bmp->data;

	//�������� ������ �� gpu
	int *CountCuda;
	unsigned char*DataCuda;
	hipMalloc(&CountCuda, sizeof(int)*NT);
	hipMalloc(&DataCuda, sizeof(unsigned char) * bmp->dh.data_size);
	
	//�������� � host'� �� device
	hipMemcpy(CountCuda, count, (sizeof(int) * NT), hipMemcpyHostToDevice);
	hipMemcpy(DataCuda, imgData, sizeof(unsigned char) * bmp->dh.data_size, hipMemcpyHostToDevice);

	//����� ���� � 7 ������
	cudaFunc<<<1, 7>>>(DataCuda, bmp->dh.data_size, NT, N, CountCuda);

	//�������� � device �� host
	hipMemcpy(count, CountCuda, sizeof(int) * NT, hipMemcpyDeviceToHost);

	int sum = 0;
	for (int i = 0; i < NT; i++) {
		sum += count[i];
	}
	printf("CUDA: %d\n", sum);

	//����������� ������
	free(imgData);
	free(count);
	freeBMP(bmp);
	hipFree(CountCuda);
	hipFree(DataCuda);
	return 0;
}
